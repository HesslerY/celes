#include "hip/hip_runtime.h"
#include "wigner_D_CUDA.cuh"
#include "translation_coefficients_CUDA.cuh"


__device__ void inner_loop(int const m, int const l2, 
                           int const s1, int const s2, int const NS,
						   float const phi, float const cosTheta,
						   float const kz,
						   float const *real_A, float const *imag_A, 
						   float const *real_B, float const *imag_B,
						   float const *real_x, float const *imag_x,
						   float *real_Wx, float *imag_Wx)
{	
	for (int m2=-l2; m2<=l2; m2++){
		float real_D1_array[LMAX+1];
		float imag_D1_array[LMAX+1];
		wignerD(m2, m, phi, cosTheta, 0, real_D1_array, imag_D1_array);
		float real_D1 = real_D1_array[l2];
		float imag_D1 = imag_D1_array[l2];									
	
		for (int m1=-LMAX; m1<=LMAX; m1++){
			float real_D2_array[LMAX+1];
			float imag_D2_array[LMAX+1];
			wignerD(m, m1, 0, cosTheta, -phi, real_D2_array, imag_D2_array);
			
			for (int l1=max(1, max(abs(m1),abs(m))); l1<=LMAX; l1++){
				float real_D2 = real_D2_array[l1] * powf(-1.0f, m+m1);  // [Doicu, B.41]
				float imag_D2 = imag_D2_array[l1] * powf(-1.0f, m+m1);
				
				for (int tau1=1; tau1<=2; tau1++){
					int n1 = (tau1-1)*LMAX*(LMAX+2)+(l1-1)*(l1+1)+l1+1+m1;
					int WxIdx = (n1-1)*NS+s1-1;
					float re_incr = 0.0f;
					float im_incr = 0.0f;
				
					for (int tau2=1; tau2<=2; tau2++){
						int n2 = (tau2-1)*LMAX*(LMAX+2)+(l2-1)*(l2+1)+l2+1+m2;
						int xIdx = (n2-1)*NS+s2-1;
						float re_xTmp = real_x[xIdx];
						float im_xTmp = imag_x[xIdx];
				
						float real_AB;
						float imag_AB;
						
						if (tau1==tau2){
							real_AB = real_A[l1];
							imag_AB = imag_A[l1];
						}
						else if (m<0){
							real_AB = -real_B[l1];
							imag_AB = -imag_B[l1];
						} 
						else {
							real_AB = real_B[l1];
							imag_AB = imag_B[l1];
						}
						
						float real_D1AB = real_D1*real_AB - imag_D1*imag_AB;
						float imag_D1AB = real_D1*imag_AB + imag_D1*real_AB;
						
						float real_D1ABD2 = real_D1AB * real_D2 - imag_D1AB * imag_D2;
						float imag_D1ABD2 = real_D1AB * imag_D2 + imag_D1AB * real_D2;
						
						re_incr += real_D1ABD2 * re_xTmp - imag_D1ABD2 * im_xTmp; 
						im_incr += real_D1ABD2 * im_xTmp + imag_D1ABD2 * re_xTmp;
					}
					
					real_Wx[WxIdx] += re_incr;
					imag_Wx[WxIdx] += im_incr;
				}
			}
		}
	}
}


__global__ void translationMatrixProduct(int const s2, int const NS, float const *sPosArray,
										 float const *sphericalBesselTable, float const *sphericalNeumannTable, float rResol,
										 float const *real_x, float const *imag_x, float *real_Wx, float *imag_Wx)
{
	//NOTE: dimensionless positions, i.e. k*xyz!!!
	int const s1 = blockDim.x * blockIdx.x + threadIdx.x + 1; // receiving sphere number (1...NS)
	float x21, y21, z21;
	float r, cosTheta, phi;
	
	if ((s1!=s2)&&s1<=NS)
	{
		// relative position
		x21 = sPosArray[3*(s1-1)]-sPosArray[3*(s2-1)];
		y21 = sPosArray[3*(s1-1)+1]-sPosArray[3*(s2-1)+1];
		z21 = sPosArray[3*(s1-1)+2]-sPosArray[3*(s2-1)+2];
		r = sqrtf(x21*x21+y21*y21+z21*z21);
		cosTheta = z21/r;
		phi = atan2f(y21,x21);

		// matrix vector product W*x -- remember that W is the transpose of A, such that indices are interchanged (l1<->l2 etc.)
				
		float real_A[LMAX+1] = {0.0f};  // SVWF axial translation coefficients, idx: l' (corresponding to l1)
		float imag_A[LMAX+1] = {0.0f};
		float real_B[LMAX+1] = {0.0f};
		float imag_B[LMAX+1] = {0.0f};
		float real_C[2*LMAX+2] = {0.0f};  // scalar SWF axial translation coefficients, idx: l' (corresponding to l1)
		float imag_C[2*LMAX+2] = {0.0f};  
		float real_C_mminus1[2*LMAX+2] = {0.0f};  // C_{m-1 l, m-1 l'}
		float imag_C_mminus1[2*LMAX+2] = {0.0f};  
		float real_C_lminus1[2*LMAX+2] = {0.0f};  // C_{m l-1, m l'}
		float imag_C_lminus1[2*LMAX+2] = {0.0f};
		float real_C_lminus2[2*LMAX+2] = {0.0f};  // C_{m l-2, m l'}
		float imag_C_lminus2[2*LMAX+2] = {0.0f};
		
		// treat m=0, l=0 in advance
		C00lprime(r, sphericalBesselTable, sphericalNeumannTable, rResol, real_C, imag_C);
		copy(real_C, real_C_mminus1);
		copy(imag_C, imag_C_mminus1);
		
		// the loop order is: m, l2, m2, m1, l1, tau1, tau2
		// reason for this twisted order:
		// m and l2 are needed for the iterative calculation of A,B
		// m and m2 are needed for the calculation of D1
		// m and m1 are needed for the calculation of D2
		for (int absm=0; absm<=LMAX; absm++){
			for (int l2=max(1,absm); l2<=LMAX; l2++){
				// compute axial translation coefficients A, B
				// l2 corresponds to l, whereas l1 corresponds to lprime
				if (l2==absm){
					Cmmlprime(absm, real_C_mminus1, imag_C_mminus1, real_C, imag_C);
					copy(real_C, real_C_mminus1);
					copy(imag_C, imag_C_mminus1);
					AB(absm, absm, r, real_C, imag_C, real_A, imag_A, real_B, imag_B);
					for (int lprime=0; lprime<2*LMAX+2; lprime++){ // reset C_{m l-1, m l'}
						real_C_lminus1[lprime] = 0.0f;
						imag_C_lminus1[lprime] = 0.0f;
					}
				} else {  // l2>absm
					copy(real_C_lminus1, real_C_lminus2);
					copy(imag_C_lminus1, imag_C_lminus2);
					copy(real_C, real_C_lminus1);
					copy(imag_C, imag_C_lminus1);
					Cmllprime(absm, l2, 
							  real_C_lminus2, imag_C_lminus2,
							  real_C_lminus1, imag_C_lminus1, 
							  real_C, imag_C);
					AB(absm, l2, r, real_C, imag_C, real_A, imag_A, real_B, imag_B);
				}
				// end of A, B computation
				
				if (absm==0) inner_loop(absm, l2, s1, s2, NS, phi, cosTheta, r, 
				                        real_A, imag_A, real_B, imag_B, 
										real_x, imag_x, real_Wx, imag_Wx);
				
				else {
					for (int m=-absm; m<=absm; m+=2*absm){
						inner_loop(m, l2, s1, s2, NS, phi, cosTheta, r, 
						           real_A, imag_A, real_B, imag_B, 
								   real_x, imag_x, real_Wx, imag_Wx);
					}
				}
			}
		}
	}
}


__global__ void translationMatrixProduct_precomputeAB(int const s2, int const NS, float const *sPosArray,
										              float const *sphericalBesselTable, float const *sphericalNeumannTable, float rResol,
													  float const *re_x, float const *im_x, float *re_Wx, float *im_Wx)
{
	//NOTE: dimensionless positions, i.e. k*xyz!!!
	int const s1 = blockDim.x * blockIdx.x + threadIdx.x + 1; // receiving sphere number (1...NS)
	float x21, y21, z21;
	float r, cosTheta, phi;
	
	float real_A[(LMAX+1)*(LMAX+1)*(LMAX+1)];  // TODO: more economic indexing
	float imag_A[(LMAX+1)*(LMAX+1)*(LMAX+1)];  
	float real_B[(LMAX+1)*(LMAX+1)*(LMAX+1)];  
	float imag_B[(LMAX+1)*(LMAX+1)*(LMAX+1)];  
	
	if ((s1!=s2)&&s1<=NS)
	{
		// relative position
		x21 = sPosArray[3*(s1-1)]-sPosArray[3*(s2-1)];
		y21 = sPosArray[3*(s1-1)+1]-sPosArray[3*(s2-1)+1];
		z21 = sPosArray[3*(s1-1)+2]-sPosArray[3*(s2-1)+2];
		
		r = sqrtf(x21*x21+y21*y21+z21*z21);
		cosTheta = z21/r;
		phi = atan2f(y21,x21);

		axialTranslationCoefficients(r, sphericalBesselTable, sphericalNeumannTable,
									 rResol, real_A, imag_A, real_B, imag_B);
		
		// matrix vector product W*x -- remember that W is the transpose of A (l1<->l2 etc.)
		for (int m=-LMAX; m<=LMAX; m++){
			for (int m1=-LMAX; m1<=LMAX; m1++){
				float real_D2_array[LMAX+1];
				float imag_D2_array[LMAX+1];
				wignerD(m, m1, 0, cosTheta, -phi, real_D2_array, imag_D2_array);
				
				for (int m2=-LMAX; m2<=LMAX; m2++){
					float real_D1_array[LMAX+1];
					float imag_D1_array[LMAX+1];
					wignerD(m2, m, phi, cosTheta, 0, real_D1_array, imag_D1_array);
					
					for (int l1=max(1, max(abs(m1),abs(m))); l1<=LMAX; l1++){
						float real_D2 = real_D2_array[l1] * powf(-1.0f, m+m1);  // [Doicu, B.41]
						float imag_D2 = imag_D2_array[l1] * powf(-1.0f, m+m1);

						for (int tau1=1; tau1<=2; tau1++){
							int n1 = (tau1-1)*LMAX*(LMAX+2)+(l1-1)*(l1+1)+l1+1+m1;
							int WxIdx = (n1-1)*NS+s1-1;
							float re_incr = 0.0f;
							float im_incr = 0.0f;
							
							for (int l2=max(1, max(abs(m2),abs(m))); l2<=LMAX; l2++){
								float real_D1 = real_D1_array[l2];
								float imag_D1 = imag_D1_array[l2];									
					
								for (int tau2=1; tau2<=2; tau2++){
									int n2 = (tau2-1)*LMAX*(LMAX+2)+(l2-1)*(l2+1)+l2+1+m2;
									int xIdx = (n2-1)*NS+s2-1;
									float re_xTmp = re_x[xIdx];
									float im_xTmp = im_x[xIdx];
							
									// axial translation
									float real_AB;
									float imag_AB;
									
									if (tau1==tau2){
										real_AB = real_A[idx(abs(m),l2,l1)];
										imag_AB = imag_A[idx(abs(m),l2,l1)];
									}
									else if (m<0){
										real_AB = -real_B[idx(-m,l2,l1)];
										imag_AB = -imag_B[idx(-m,l2,l1)];
									} 
									else {
										real_AB = real_B[idx(m, l2, l1)];
										imag_AB = imag_B[idx(m, l2, l1)];
									}
									
									float real_D1AB = real_D1*real_AB - imag_D1*imag_AB;
									float imag_D1AB = real_D1*imag_AB + imag_D1*real_AB;
									
									float real_D1ABD2 = real_D1AB * real_D2 - imag_D1AB * imag_D2;
									float imag_D1ABD2 = real_D1AB * imag_D2 + imag_D1AB * real_D2;
									
									re_incr += real_D1ABD2 * re_xTmp - imag_D1ABD2 * im_xTmp; 
									im_incr += real_D1ABD2 * im_xTmp + imag_D1ABD2 * re_xTmp;
								}
									//if (n2==1){
									//	re_Wx[WxIdx] = real_D2;
									//	im_Wx[WxIdx] = imag_D2;
									//}
							}
							re_Wx[WxIdx] += re_incr;
							im_Wx[WxIdx] += im_incr;
						}
					}
				} 
			}  
		}
	} 
}
