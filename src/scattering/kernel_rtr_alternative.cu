#include "hip/hip_runtime.h"
#include "wigner_D_CUDA.cuh"
#include "translation_coefficients_CUDA.cuh"


__global__ void translationMatrixProduct(int const s2, int const NS, float const *sPosArray,
										 float const *sphericalBesselTable, float const *sphericalNeumannTable, float rResol,
										 float const *re_x, float const *im_x, float *re_Wx, float *im_Wx)
{
	//NOTE: dimensionless positions, i.e. k*xyz!!!
	int const s1 = blockDim.x * blockIdx.x + threadIdx.x + 1; // receiving sphere number (1...NS)
	float x21, y21, z21;
	float r, cosTheta, phi;
	
	float real_A[(LMAX+1)*(LMAX+1)*(LMAX+1)];  // TODO: more economic indexing
	float imag_A[(LMAX+1)*(LMAX+1)*(LMAX+1)];  
	float real_B[(LMAX+1)*(LMAX+1)*(LMAX+1)];  
	float imag_B[(LMAX+1)*(LMAX+1)*(LMAX+1)];  
	
	if ((s1!=s2)&&s1<=NS)
	{
		// relative position
		x21 = sPosArray[3*(s1-1)]-sPosArray[3*(s2-1)];
		y21 = sPosArray[3*(s1-1)+1]-sPosArray[3*(s2-1)+1];
		z21 = sPosArray[3*(s1-1)+2]-sPosArray[3*(s2-1)+2];
		
		r = sqrtf(x21*x21+y21*y21+z21*z21);
		cosTheta = z21/r;
		phi = atan2f(y21,x21);

		axialTranslationCoefficients(r, sphericalBesselTable, sphericalNeumannTable,
									 rResol, real_A, imag_A, real_B, imag_B);
		
		// matrix vector product W*x -- remember that W is the transpose of A (l1<->l2 etc.)
		for (int m=-LMAX; m<=LMAX; m++){
			for (int m1=-LMAX; m1<=LMAX; m1++){
				float real_D2_array[LMAX+1];
				float imag_D2_array[LMAX+1];
				wignerD(m, m1, 0, cosTheta, -phi, real_D2_array, imag_D2_array);
				
				for (int m2=-LMAX; m2<=LMAX; m2++){
					float real_D1_array[LMAX+1];
					float imag_D1_array[LMAX+1];
					wignerD(m2, m, phi, cosTheta, 0, real_D1_array, imag_D1_array);
					
					for (int l1=max(1, max(abs(m1),abs(m))); l1<=LMAX; l1++){
						float real_D2 = real_D2_array[l1] * powf(-1.0f, m+m1);  // [Doicu, B.41]
						float imag_D2 = imag_D2_array[l1] * powf(-1.0f, m+m1);

						for (int tau1=1; tau1<=2; tau1++){
							int n1 = (tau1-1)*LMAX*(LMAX+2)+(l1-1)*(l1+1)+l1+1+m1;
							int WxIdx = (n1-1)*NS+s1-1;
							float re_incr = 0.0f;
							float im_incr = 0.0f;
							
							for (int l2=max(1, max(abs(m2),abs(m))); l2<=LMAX; l2++){
								float real_D1 = real_D1_array[l2];
								float imag_D1 = imag_D1_array[l2];									
					
								for (int tau2=1; tau2<=2; tau2++){
									int n2 = (tau2-1)*LMAX*(LMAX+2)+(l2-1)*(l2+1)+l2+1+m2;
									int xIdx = (n2-1)*NS+s2-1;
									float re_xTmp = re_x[xIdx];
									float im_xTmp = im_x[xIdx];
							
									// axial translation
									float real_AB;
									float imag_AB;
									
									if (tau1==tau2){
										real_AB = real_A[idx(abs(m),l2,l1)];
										imag_AB = imag_A[idx(abs(m),l2,l1)];
									}
									else if (m<0){
										real_AB = -real_B[idx(-m,l2,l1)];
										imag_AB = -imag_B[idx(-m,l2,l1)];
									} 
									else {
										real_AB = real_B[idx(m, l2, l1)];
										imag_AB = imag_B[idx(m, l2, l1)];
									}
									
									float real_D1AB = real_D1*real_AB - imag_D1*imag_AB;
									float imag_D1AB = real_D1*imag_AB + imag_D1*real_AB;
									
									float real_D1ABD2 = real_D1AB * real_D2 - imag_D1AB * imag_D2;
									float imag_D1ABD2 = real_D1AB * imag_D2 + imag_D1AB * real_D2;
									
									re_incr += real_D1ABD2 * re_xTmp - imag_D1ABD2 * im_xTmp; 
									im_incr += real_D1ABD2 * im_xTmp + imag_D1ABD2 * re_xTmp;
								}
									//if (n2==1){
									//	re_Wx[WxIdx] = real_D2;
									//	im_Wx[WxIdx] = imag_D2;
									//}
							}
							re_Wx[WxIdx] += re_incr;
							im_Wx[WxIdx] += im_incr;
						}
					}
				} 
			}  
		}
	} 
}
