#include "gpu/mxGPUArray.h"
#include "mex.h"
#include "kernel_rtr.cu"

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128
#endif


/*=============================================================================
@brief 	Interface to Matlab: coupling matrix multiply W*x
=============================================================================*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	/* input arguments: 
	real_x
	imag_x 
	real_h			
	imag_h	
	spos
	NS 			
	rResol */
	
	// initialize the MathWorks GPU API.
	mxInitGPU();
	
	// check number of arguments:
	if (nrhs!=7) {mexErrMsgTxt("wrong number of input arguments");}
	if (nlhs!=2) {mexErrMsgTxt("wrong number of output arguments");}
	
	// check for GPUArrays
	if (!(mxIsGPUArray(prhs[0]))) {mexErrMsgTxt("real_x is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[1]))) {mexErrMsgTxt("imag_x is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[2]))) {mexErrMsgTxt("real_h is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[3]))) {mexErrMsgTxt("imag_h is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[4]))) {mexErrMsgTxt("spos is not a gpuArray");}
	
	// initialize mxGPUArrays
	mxGPUArray const *mx_real_x = mxGPUCreateFromMxArray(prhs[0]);	
	mxGPUArray const *mx_imag_x = mxGPUCreateFromMxArray(prhs[1]);	
	mxGPUArray const *mx_real_h = mxGPUCreateFromMxArray(prhs[2]);	
	mxGPUArray const *mx_imag_h = mxGPUCreateFromMxArray(prhs[3]);	
	mxGPUArray const *mx_sPos = mxGPUCreateFromMxArray(prhs[4]);
	
	mxGPUArray *mx_real_Wx = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(mx_real_x),
                             mxGPUGetDimensions(mx_real_x),
                             mxSINGLE_CLASS,mxREAL,MX_GPU_INITIALIZE_VALUES);
																
	mxGPUArray *mx_imag_Wx = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(mx_real_x),
                             mxGPUGetDimensions(mx_real_x),
                             mxSINGLE_CLASS,mxREAL,MX_GPU_INITIALIZE_VALUES);

	// check data types
	if (mxGPUGetClassID(mx_real_x) != mxSINGLE_CLASS) {mexErrMsgTxt("real_x is not single");}
	if (mxGPUGetClassID(mx_imag_x) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_x is not single");}
	if (mxGPUGetClassID(mx_real_h) != mxSINGLE_CLASS) {mexErrMsgTxt("real_h is not single");}
	if (mxGPUGetClassID(mx_imag_h) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_h is not single");}
	if (mxGPUGetClassID(mx_sPos) != mxSINGLE_CLASS) {mexErrMsgTxt("sPos is not single");}
	if (mxGetClassID(prhs[5]) != mxINT32_CLASS) {mexErrMsgTxt("NS is not int32");}
	if (mxGetClassID(prhs[6]) != mxSINGLE_CLASS) {mexErrMsgTxt("rResol is not single");}

	// initialize host variables
	int const *NS = (int*)mxGetData(prhs[5]);			// total number of spheres
	float const *rResol = (float*)mxGetData(prhs[6]);	// resolution of lookups
	int const blocksPerGrid = (NS[0] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	
	// initialize device variables
	float const *d_real_x = (float const *)(mxGPUGetDataReadOnly(mx_real_x));
	float const *d_imag_x = (float const *)(mxGPUGetDataReadOnly(mx_imag_x));
	float const *d_real_h = (float const *)(mxGPUGetDataReadOnly(mx_real_h));
	float const *d_imag_h = (float const *)(mxGPUGetDataReadOnly(mx_imag_h));
	float const *d_sPos = (float const *)(mxGPUGetDataReadOnly(mx_sPos));
	float	 	*d_real_Wx = (float *)(mxGPUGetData(mx_real_Wx));
	float	 	*d_imag_Wx = (float *)(mxGPUGetData(mx_imag_Wx));
	
	// start computation
	for (int s2=1; s2<=NS[0]; s2++)
	{
		translationMatrixProduct<<< blocksPerGrid, THREADS_PER_BLOCK >>> (s2, NS[0], d_sPos,
																		  d_real_h, d_imag_h, rResol[0],
																		  d_real_x, d_imag_x, 
																		  d_real_Wx, d_imag_Wx);
																		
		//mexPrintf("sphere %i\n",s2);
		// hipMemcpy(&check_re,d_real_Wx,sizeof(check_re),hipMemcpyDeviceToHost);
		// hipMemcpy(&check_im,d_imag_Wx,sizeof(check_im),hipMemcpyDeviceToHost);
		// mexPrintf("%f %f\n",check_re,check_im);
	}

	// wrap the result up as a MATLAB gpuArray for return
	plhs[0] = mxGPUCreateMxArrayOnGPU(mx_real_Wx);
	plhs[1] = mxGPUCreateMxArrayOnGPU(mx_imag_Wx);
	
	// destroy mxgpuarrays
	mxGPUDestroyGPUArray(mx_real_x);
	mxGPUDestroyGPUArray(mx_imag_x);
	mxGPUDestroyGPUArray(mx_real_h);
	mxGPUDestroyGPUArray(mx_imag_h);
	mxGPUDestroyGPUArray(mx_sPos);
	mxGPUDestroyGPUArray(mx_real_Wx);
	mxGPUDestroyGPUArray(mx_imag_Wx);
	
	//hipDeviceReset();  // necessary for profiling
}
