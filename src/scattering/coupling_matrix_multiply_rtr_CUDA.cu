#include "hip/hip_runtime.h"
#include "gpu/mxGPUArray.h"
#include "mex.h"
#include "wigner_D_CUDA.cuh"
#include "translation_coefficients_CUDA.cuh"

/*=============================================================================
@brief 	Evaluate the lookup of the spherical Hankel function with cubic 
		spline interpolation

@param	p 			Spherical Hankel function order
@param	r			Radial position
@param	spjTable	Pointer to lookup table
@param	rResol		Sampling distance of radial position

@retval 			Interpolated value of spherical Hankel function
=============================================================================*/
__device__ float interpolateHankelLookup(int const p, float const r, float const *spjTable, float const rResol)
{
	float spj;
	float rPos = r/rResol;
	int rIdx = int(rPos);    					// points to table position -1, because for each p, the first entry with respect to r in the spjTable is copied 
	rPos -= rIdx; 							 	// (remainder of r/rResol) / rResol
	float rPos2 = pow(rPos,2);
	float rPos3 = pow(rPos,3);
	spj = ((-rPos3+2*rPos2-rPos) * spjTable[rIdx*(2*LMAX+1)+p]
			+ (3*rPos3-5*rPos2+2) * spjTable[(rIdx+1)*(2*LMAX+1)+p]
			+ (-3*rPos3+4*rPos2+rPos) * spjTable[(rIdx+2)*(2*LMAX+1)+p]
			+ (rPos3-rPos2) * spjTable[(rIdx+3)*(2*LMAX+1)+p])/2;
	return spj;
}


__global__ void testWignerD(int const m, int const m_prime, 
                            float const alpha, float const cosBeta, float const gamma, 
							float *real_D, float *imag_D)
{
	wignerD(m, m_prime, alpha, cosBeta, gamma, real_D, imag_D);
}


__global__ void testAxialTranslation(const float kz,
								     float const *jTable, float const *yTable,
								     const float rResol,
								     float *real_A, float *imag_A, 
								     float *real_B, float *imag_B)
{
	axialTranslationCoefficients(kz, jTable, yTable, rResol, 
	                             real_A, imag_A, real_B, imag_B);
}

/*=============================================================================
@brief 	Interface to Matlab

=============================================================================*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	/* input arguments: 
	real_x
	imag_x 
	real_h			
	imag_h	
	Plm_coeffs
	real_ab5
	imag_ab5
	spos
	NS 			
	rResol */
	
	/*
	m, m_prime, alpha, cosBeta, gamma, kz, jTable, yTable, rResol
	*/
	
	
	// initialize the MathWorks GPU API.
	mxInitGPU();
	
	/*
	// check number of arguments:
	if (nrhs!=10) {mexErrMsgTxt("wrong number of input arguments");}
	if (nlhs!=2) {mexErrMsgTxt("wrong number of output arguments");}

	// check for GPUArrays
	if (!(mxIsGPUArray(prhs[0]))) {mexErrMsgTxt("real_x is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[1]))) {mexErrMsgTxt("imag_x is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[2]))) {mexErrMsgTxt("real_h is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[3]))) {mexErrMsgTxt("imag_h is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[4]))) {mexErrMsgTxt("Plm_coeffs is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[5]))) {mexErrMsgTxt("real_ab5 is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[6]))) {mexErrMsgTxt("imag_ab5 is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[7]))) {mexErrMsgTxt("spos is not a gpuArray");}
	
	
	// initialize mxGPUArrays
	mxGPUArray const *mx_real_x = mxGPUCreateFromMxArray(prhs[0]);	
	mxGPUArray const *mx_imag_x = mxGPUCreateFromMxArray(prhs[1]);	
	mxGPUArray const *mx_real_h = mxGPUCreateFromMxArray(prhs[2]);	
	mxGPUArray const *mx_imag_h = mxGPUCreateFromMxArray(prhs[3]);	
	mxGPUArray const *mx_PlmCoeff = mxGPUCreateFromMxArray(prhs[4]);	
	mxGPUArray const *mx_real_ab5 = mxGPUCreateFromMxArray(prhs[5]);
	mxGPUArray const *mx_imag_ab5 = mxGPUCreateFromMxArray(prhs[6]);
	mxGPUArray const *mx_sPos = mxGPUCreateFromMxArray(prhs[7]);
	// mxGPUArray *mx_real_Wx = mxGPUCopyGPUArray(mx_real_x);  // writable copy of real_x ... to be overwritten in kernel
	// mxGPUArray *mx_imag_Wx = mxGPUCopyGPUArray(mx_imag_x);
	*/
	
	mxGPUArray const *mx_jTable = mxGPUCreateFromMxArray(prhs[6]);	
	mxGPUArray const *mx_yTable = mxGPUCreateFromMxArray(prhs[7]);	
	
	const mwSize dim[1] = {LMAX+1};
	mxGPUArray *mx_real_D = mxGPUCreateGPUArray((mwSize) 1,
	                                            dim,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);

	mxGPUArray *mx_imag_D = mxGPUCreateGPUArray((mwSize) 1,
	                                            dim,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);												

	const mwSize dim3[3] = {LMAX+1, LMAX+1, LMAX+1};
	mxGPUArray *mx_real_A = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);

	mxGPUArray *mx_imag_A = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);																								
	
	mxGPUArray *mx_real_B = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);

	mxGPUArray *mx_imag_B = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);																									
	/*
	// check data types
	if (mxGPUGetClassID(mx_real_x) != mxSINGLE_CLASS) {mexErrMsgTxt("real_x is not single");}
	if (mxGPUGetClassID(mx_imag_x) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_x is not single");}
	if (mxGPUGetClassID(mx_real_h) != mxSINGLE_CLASS) {mexErrMsgTxt("real_h is not single");}
	if (mxGPUGetClassID(mx_imag_h) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_h is not single");}
	if (mxGPUGetClassID(mx_PlmCoeff) != mxSINGLE_CLASS) {mexErrMsgTxt("PlmCoeff is not single");}
	if (mxGPUGetClassID(mx_real_ab5) != mxSINGLE_CLASS) {mexErrMsgTxt("real_ab5 is not single");}
	if (mxGPUGetClassID(mx_imag_ab5) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_ab5 is not single");}
	if (mxGPUGetClassID(mx_sPos) != mxSINGLE_CLASS) {mexErrMsgTxt("sPos is not single");}
	if (mxGetClassID(prhs[8]) != mxINT32_CLASS) {mexErrMsgTxt("NS is not int32");}
	if (mxGetClassID(prhs[9]) != mxSINGLE_CLASS) {mexErrMsgTxt("rResol is not single");}
	*/
	
	// initialize host variables
	int const *m = (int*)mxGetData(prhs[0]);
	int const *m_prime = (int*)mxGetData(prhs[1]);
	float const *alpha = (float*)mxGetData(prhs[2]);
	float const *cosBeta = (float*)mxGetData(prhs[3]);
	float const *gamma = (float*)mxGetData(prhs[4]);
	float const *kz = (float*)mxGetData(prhs[5]);
	float const *rResol = (float*)mxGetData(prhs[8]);
	
	/*
	float const *rResol = (float*)mxGetData(prhs[9]);	// maximal polar quantum number
	int const 	threadsPerBlock = 256;
	int const blocksPerGrid = (NS[0] + threadsPerBlock - 1) / threadsPerBlock;
	*/
	
	// initialize device variables
	float	 	*d_real_D = (float *)(mxGPUGetData(mx_real_D));
	float	 	*d_imag_D = (float *)(mxGPUGetData(mx_imag_D));
	
	float	 	*d_real_A = (float *)(mxGPUGetData(mx_real_A));
	float	 	*d_imag_A = (float *)(mxGPUGetData(mx_imag_A));
	
	float	 	*d_real_B = (float *)(mxGPUGetData(mx_real_B));
	float	 	*d_imag_B = (float *)(mxGPUGetData(mx_imag_B));
	
	float const *d_jTable = (float const *)(mxGPUGetDataReadOnly(mx_jTable));
	float const *d_yTable = (float const *)(mxGPUGetDataReadOnly(mx_yTable));
	
	/*
	float const *d_real_x = (float const *)(mxGPUGetDataReadOnly(mx_real_x));
	float const *d_imag_x = (float const *)(mxGPUGetDataReadOnly(mx_imag_x));
	float const *d_real_h = (float const *)(mxGPUGetDataReadOnly(mx_real_h));
	float const *d_imag_h = (float const *)(mxGPUGetDataReadOnly(mx_imag_h));
	float const *d_PlmCoeff = (float const *)(mxGPUGetDataReadOnly(mx_PlmCoeff));
	float const *d_real_ab5 = (float const *)(mxGPUGetDataReadOnly(mx_real_ab5));
	float const *d_imag_ab5 = (float const *)(mxGPUGetDataReadOnly(mx_imag_ab5));
	float const *d_sPos = (float const *)(mxGPUGetDataReadOnly(mx_sPos));
	*/
	
	// float check_re;
	// float check_im;
	
	mexPrintf("m=");
	mexPrintf("%i\n", m[0]);
	mexPrintf("m'=");
	mexPrintf("%i\n", m_prime[0]);
	mexPrintf("alpha=");
	mexPrintf("%f\n", alpha[0]);
	mexPrintf("cos beta=");
	mexPrintf("%f\n", cosBeta[0]);
	mexPrintf("gamma=");
	mexPrintf("%f\n", gamma[0]);	
	
	testWignerD<<<1,1>>>(m[0], m_prime[0], alpha[0], cosBeta[0], gamma[0], d_real_D, d_imag_D);
	
	testAxialTranslation<<<1,1>>>(kz[0], d_jTable, d_yTable, rResol[0], 
	                              d_real_A, d_imag_A, d_real_B, d_imag_B);
	
	plhs[0] = mxGPUCreateMxArrayOnGPU(mx_real_D);
	plhs[1] = mxGPUCreateMxArrayOnGPU(mx_imag_D);
	plhs[2] = mxGPUCreateMxArrayOnGPU(mx_real_A);
	plhs[3] = mxGPUCreateMxArrayOnGPU(mx_imag_A);
	plhs[4] = mxGPUCreateMxArrayOnGPU(mx_real_B);
	plhs[5] = mxGPUCreateMxArrayOnGPU(mx_imag_B);
	
	/*
	// start computation
	for (int s2=1; s2<=NS[0]; s2++)
	{
		translationMatrixProduct<<< blocksPerGrid,threadsPerBlock >>> (s2,NS[0],d_sPos,
																		d_real_h, d_imag_h, rResol[0],
																		d_PlmCoeff, d_real_ab5, d_imag_ab5,
																		d_real_x, d_imag_x, d_real_Wx, d_imag_Wx);
		// hipMemcpy(&check_re,d_real_Wx,sizeof(check_re),hipMemcpyDeviceToHost);
		// hipMemcpy(&check_im,d_imag_Wx,sizeof(check_im),hipMemcpyDeviceToHost);
		// mexPrintf("%f %f\n",check_re,check_im);
																																
	}

	// wrap the result up as a MATLAB gpuArray for return
	plhs[0] = mxGPUCreateMxArrayOnGPU(mx_real_Wx);
	plhs[1] = mxGPUCreateMxArrayOnGPU(mx_imag_Wx);
	*/
	
	// destroy mxgpuarrays
	/*
	mxGPUDestroyGPUArray(mx_real_x);
	mxGPUDestroyGPUArray(mx_imag_x);
	mxGPUDestroyGPUArray(mx_real_h);
	mxGPUDestroyGPUArray(mx_imag_h);
	mxGPUDestroyGPUArray(mx_PlmCoeff);
	mxGPUDestroyGPUArray(mx_real_ab5);
	mxGPUDestroyGPUArray(mx_imag_ab5);
	mxGPUDestroyGPUArray(mx_sPos);
	mxGPUDestroyGPUArray(mx_real_Wx);
	mxGPUDestroyGPUArray(mx_imag_Wx);
	*/
	mxGPUDestroyGPUArray(mx_real_D);
	mxGPUDestroyGPUArray(mx_imag_D);
	mxGPUDestroyGPUArray(mx_real_A);
	mxGPUDestroyGPUArray(mx_imag_A);
	mxGPUDestroyGPUArray(mx_real_B);
	mxGPUDestroyGPUArray(mx_imag_B);
	//hipDeviceReset();  // necessary for profiling
}


