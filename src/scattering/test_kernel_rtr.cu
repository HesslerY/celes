#include "hip/hip_runtime.h"
#include "gpu/mxGPUArray.h"
#include "mex.h"
#include "wigner_D_CUDA.cuh"
#include "translation_coefficients_CUDA.cuh"

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 256
#endif


__global__ void testWignerD(int const m, int const m_prime, 
                            float const alpha, float const cosBeta, float const gamma, 
							float *real_D, float *imag_D)
{
	wignerD(m, m_prime, alpha, cosBeta, gamma, real_D, imag_D);
}


__global__ void testAxialTranslation(const float kz,
								     float const *jTable, float const *yTable,
								     const float rResol,
								     float *real_A, float *imag_A, 
								     float *real_B, float *imag_B)
{
	axialTranslationCoefficients(kz, jTable, yTable, rResol, 
	                             real_A, imag_A, real_B, imag_B);
}


/*=============================================================================
@brief 	Interface to Matlab: test translation coefficients and wigner_D
=============================================================================*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	/* input arguments: 
	m, m_prime, alpha, cosBeta, gamma, kz, jTable, yTable, rResol
	*/
	
	mxInitGPU(); // initialize the MathWorks GPU API.
	
	mxGPUArray const *mx_jTable = mxGPUCreateFromMxArray(prhs[6]);	
	mxGPUArray const *mx_yTable = mxGPUCreateFromMxArray(prhs[7]);	
	
	const mwSize dim[1] = {LMAX+1};
	mxGPUArray *mx_real_D = mxGPUCreateGPUArray((mwSize) 1,
	                                            dim,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);

	mxGPUArray *mx_imag_D = mxGPUCreateGPUArray((mwSize) 1,
	                                            dim,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);												

	const mwSize dim3[3] = {LMAX+1, LMAX+1, LMAX+1};
	mxGPUArray *mx_real_A = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);

	mxGPUArray *mx_imag_A = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);																								
	
	mxGPUArray *mx_real_B = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);

	mxGPUArray *mx_imag_B = mxGPUCreateGPUArray((mwSize) 3,
	                                            dim3,
												mxSINGLE_CLASS,
												mxREAL,
												MX_GPU_INITIALIZE_VALUES);																									
	
	// initialize host variables
	int const *m = (int*)mxGetData(prhs[0]);
	int const *m_prime = (int*)mxGetData(prhs[1]);
	float const *alpha = (float*)mxGetData(prhs[2]);
	float const *cosBeta = (float*)mxGetData(prhs[3]);
	float const *gamma = (float*)mxGetData(prhs[4]);
	float const *kz = (float*)mxGetData(prhs[5]);
	float const *rResol = (float*)mxGetData(prhs[8]);
	
	// initialize device variables
	float	 	*d_real_D = (float *)(mxGPUGetData(mx_real_D));
	float	 	*d_imag_D = (float *)(mxGPUGetData(mx_imag_D));
	
	float	 	*d_real_A = (float *)(mxGPUGetData(mx_real_A));
	float	 	*d_imag_A = (float *)(mxGPUGetData(mx_imag_A));
	
	float	 	*d_real_B = (float *)(mxGPUGetData(mx_real_B));
	float	 	*d_imag_B = (float *)(mxGPUGetData(mx_imag_B));
	
	float const *d_jTable = (float const *)(mxGPUGetDataReadOnly(mx_jTable));
	float const *d_yTable = (float const *)(mxGPUGetDataReadOnly(mx_yTable));
	
	mexPrintf("m=");
	mexPrintf("%i\n", m[0]);
	mexPrintf("m'=");
	mexPrintf("%i\n", m_prime[0]);
	mexPrintf("alpha=");
	mexPrintf("%f\n", alpha[0]);
	mexPrintf("cos beta=");
	mexPrintf("%f\n", cosBeta[0]);
	mexPrintf("gamma=");
	mexPrintf("%f\n", gamma[0]);	
	
	testWignerD<<<1,1>>>(m[0], m_prime[0], alpha[0], cosBeta[0], gamma[0], d_real_D, d_imag_D);
	
	testAxialTranslation<<<1,1>>>(kz[0], d_jTable, d_yTable, rResol[0], 
	                              d_real_A, d_imag_A, d_real_B, d_imag_B);
	
	plhs[0] = mxGPUCreateMxArrayOnGPU(mx_real_D);
	plhs[1] = mxGPUCreateMxArrayOnGPU(mx_imag_D);
	plhs[2] = mxGPUCreateMxArrayOnGPU(mx_real_A);
	plhs[3] = mxGPUCreateMxArrayOnGPU(mx_imag_A);
	plhs[4] = mxGPUCreateMxArrayOnGPU(mx_real_B);
	plhs[5] = mxGPUCreateMxArrayOnGPU(mx_imag_B);
	
	mxGPUDestroyGPUArray(mx_real_D);
	mxGPUDestroyGPUArray(mx_imag_D);
	mxGPUDestroyGPUArray(mx_real_A);
	mxGPUDestroyGPUArray(mx_imag_A);
	mxGPUDestroyGPUArray(mx_real_B);
	mxGPUDestroyGPUArray(mx_imag_B);
	//hipDeviceReset();  // necessary for profiling
}
